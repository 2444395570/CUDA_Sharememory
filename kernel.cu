#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <memory>

using namespace std;

__global__ void gpu_shared_memory(float* d_a) {
	int i, index = threadIdx.x;
	float average, sum = 0.0f;
	//���干���ڴ�
	__shared__ float sh_arr[10];
	sh_arr[index] = d_a[index];

	//����ָ��ȷ�����ڹ����ڴ������д������Ѿ���ɡ�
	/*
	__syncthreads()��cuda���ڽ����������ڿ����߳�ͨ��.��Щ���Ե���__syncthreads()���߳���Ҫ�������Ե���
	�õ���̣߳������ǵȴ��������������̡߳�
	*/
	__syncthreads();

	for (int i = 0; i <= index; i++)
	{
		sum += sh_arr[i];
	}
	average = sum / (index + 1.0f);
	d_a[index] = average;


	sh_arr[index] = average;
	//���д����Ƕ���ģ����ҽ������������ִ��û���κ�Ӱ�졣���һ�д��뽫�����ŵ��˹����ڴ��С����д���
	//������ִ����˵û��Ӱ�졣��Ϊ�����ڴ�������ڵ���ǰ��ִ����Ͼͽ����ˡ�
}

int main(int argc, char** argv) {
	float h_a[10];
	float* d_a;

	//��ʼ����������
	for (int i = 0; i < 10; i++)
	{
		h_a[i] = i;
	}

	//����ȫ���ڴ浽�豸��
	hipMalloc((void**)&d_a, sizeof(float) * 10);
	//�������ڴ浽�豸�ڴ��ϸ�������
	hipMemcpy((void*)d_a, (void*)h_a, sizeof(float) * 10,hipMemcpyHostToDevice);
	gpu_shared_memory << <1, 10 >> > (d_a);
	//�����޸Ĺ������鷵�ص�����
	hipMemcpy((void*)h_a, (void*)d_a, sizeof(float) * 10, hipMemcpyDeviceToHost);
	printf("Use of shared Memory on GPU:\n");
	for (int i = 0; i < 10; i++)
	{
		printf("The running average after %d element is %f\n", i, h_a[i]);
	}
	return 0;
}